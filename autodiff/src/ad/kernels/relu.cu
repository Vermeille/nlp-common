#include "hip/hip_runtime.h"
#include "kernels.h"

namespace ad {
namespace cuda {

__global__
void cuRelu(float* res, const float* arr1, size_t sz) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    for (; i < sz; i += blockDim.x * gridDim.x) {
        res[i] = (arr1[i] > 0) ? arr1[i] : 0;
    }
}

void Relu(::cuda::Ptr<float> res, const ::cuda::Ptr<float> arr1, size_t sz) {
    cuRelu<<<(sz + 128 - 1) / 128, 128>>>(res.Get(), arr1.Get(), sz);
    hipDeviceSynchronize();
}

__global__
void cuReluGrad(float* res, const float* arr1, size_t sz) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    for (; i < sz; i += blockDim.x * gridDim.x) {
        res[i] = (arr1[i] > 0) ? 1 : 0;
    }
}

void ReluGrad(::cuda::Ptr<float> res, const ::cuda::Ptr<float> arr1, size_t sz) {
    cuReluGrad<<<(sz + 128 - 1) / 128, 128>>>(res.Get(), arr1.Get(), sz);
    hipDeviceSynchronize();
}

} // cuda
} // ad
