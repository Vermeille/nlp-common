#include "hip/hip_runtime.h"
#include "initializers.h"
#include "helpers.h"

#include "kernels/kernels.h"

#include <cmath>
#include <hiprand.h>

namespace ad {

static int i = 0;

static hiprandGenerator_t* GetUniformGenerator() {
    static hiprandGenerator_t* gen = nullptr;
    if (!gen) {
        gen = new hiprandGenerator_t;
        hiprandCreateGenerator(gen, HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(*gen, 1234ULL);

        std::atexit([&]() {
                hiprandDestroyGenerator(*gen);
            });
    }
    return gen;
}

void Gaussian::Init(Matrix& mat) const {
    hiprandGenerator_t* gen = GetUniformGenerator();
    hiprandGenerateNormal(*gen, mat.data().Get(), mat.size(), mu_, sigma_);
    std::cout << "gaussian inited " << i << "\n";
    ++i;
}

void Uniform::Init(Matrix& mat) const {
    hiprandGenerator_t* gen = GetUniformGenerator();
    hiprandGenerateUniform(*gen, mat.data().Get(), mat.size());
    auto x = cuda::Array(mat.data());
    auto scale = cuda::Value(std::fabs(from_) + std::fabs(to_));
    auto from = cuda::Value(from_);
    cuda::RunKernel(cuda::Seq(
            x = x * scale + from),
        mat.size());
    std::cout << "uniform inited " << i << "\n";
    ++i;
}

} // ad
